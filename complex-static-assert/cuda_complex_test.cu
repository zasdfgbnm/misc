
#include <hip/hip_runtime.h>
template<typename T>
struct alignas(sizeof(T) * 2) complex {
  using value_type = T;

  T real_ = T(0);
  T imag_ = T(0);

  constexpr complex(const T& re, const T& im = T()): real_(re), imag_(im) {}

  template<typename U>
  constexpr complex<T> &operator -=(const complex<U> &rhs) {
    real_ -= rhs.real_;
    imag_ -= rhs.imag_;
    return *this;
  }
};

template<typename scalar_t, typename rhs_t>
constexpr complex<scalar_t> m(scalar_t real, scalar_t imag, complex<rhs_t> rhs) {
  complex<scalar_t> result(real, imag);
  result -= rhs;
  return result;
}

__global__ void test_arithmetic_assign() {
  constexpr complex<float> y3 = m(float(2), float(2), complex<double>(0.0, 1.0));
  static_assert(y3.real_ == float(2), "");
}

int main() {}