#include <iostream>
#include <hipcub/hipcub.hpp>
#include <chrono>

using namespace std::chrono;

constexpr size_t N = 1000000;

int main() {
    int *h1, *h2, *d1, *d2, *d3, *d4, *tmp;
    h1 = new int[N];
    h2 = new int[N];
    for(size_t i = 0; i < N; i++) {
        h1[i] = N - i;
    }
    hipMalloc(&d1, sizeof(int) * N);
    hipMalloc(&d2, sizeof(int) * N);
    hipMalloc(&d3, sizeof(int) * N);
    hipMalloc(&d4, sizeof(int) * N);
    hipMemcpy(d1, h1, sizeof(int) * N, hipMemcpyDefault);

    size_t temp_storage_bytes;
    hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes, d1, d2, d3, d4, N);
    hipMalloc(&tmp, temp_storage_bytes);

    hipDeviceSynchronize();
    auto start = high_resolution_clock::now();
    hipcub::DeviceRadixSort::SortPairs(tmp, temp_storage_bytes, d1, d2, d3, d4, N);
    hipDeviceSynchronize();
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    std::cout << duration.count() << std::endl;

    hipMemcpy(h2, d2, sizeof(int) * N, hipMemcpyDefault);
    for(size_t i = 0; i < 10; i++) {
        std::cout << h2[i] << ", ";
    }
}